
#include <hip/hip_runtime.h>
#include <iostream>           // Printing
#include <vector>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMV
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <hipblas.h>

#define DEFAULT_FPX double
#if (__cplusplus >= 201703L)  ///< if c++17 or above
#define TEMPLATE_WITH_TYPE_T template<typename T = DEFAULT_FPX>
#else
#define TEMPLATE_WITH_TYPE_T template<typename T>
#endif
//
//
///* ================================================================================================
// *  ERROR CHECKING
// * ================================================================================================ */

/**
 * Check for errors when calling GPU functions
 */
#define gpuErrChk(status) { gpuAssert((status), __FILE__, __LINE__); } while(false)

TEMPLATE_WITH_TYPE_T inline void gpuAssert(T code, const char *file, int line, bool abort = true) {
    if constexpr (std::is_same_v<T, hipError_t>) {
        if (code != hipSuccess) {
            std::cerr << "cuda error. String: " << hipGetErrorString(code)
                      << ", file: " << file << ", line: " << line << "\n";
            if (abort) exit(code);
        }
    } else if constexpr (std::is_same_v<T, hipblasStatus_t>) {
        if (code != HIPBLAS_STATUS_SUCCESS) {
            std::cerr << "cublas error. Name: " << cublasGetStatusName(code)
                      << ", string: " << cublasGetStatusString(code)
                      << ", file: " << file << ", line: " << line << "\n";
            if (abort) exit(code);
        }
    } else if constexpr(std::is_same_v<T, hipsparseStatus_t>) {
        if (code != HIPSPARSE_STATUS_SUCCESS) {
            std::cerr << "cublas error. Code: " << hipsparseGetErrorString(code)
                      << ", file: " << file << ", line: " << line << "\n";
            if (abort) exit(code);
        }
    } else {
        std::cerr << "Error: library status parser not implemented" << "\n";
    }
}



/* ================================================================================================
 *  SESSION
 * ================================================================================================ */
/**
 * Singleton for Cuda library handles
 */
class Session {
public:

    static Session &getInstance() {
        static Session instance;
        return instance;
    }

private:
    Session() {
        gpuErrChk(hipblasCreate(&m_cublasHandle));
        gpuErrChk(hipsparseCreate(&m_sparseHandle));
    }

    ~Session() {
        gpuErrChk(hipblasDestroy(m_cublasHandle));
        gpuErrChk(hipsparseDestroy(m_sparseHandle));
    }

    hipblasHandle_t m_cublasHandle;
    hipsparseHandle_t m_sparseHandle;


public:
    Session(Session const &) = delete;

    void operator=(Session const &) = delete;

    hipblasHandle_t &cuBlasHandle() { return m_cublasHandle; }

    hipsparseHandle_t &cuSpraseHandle() { return m_sparseHandle; }
};


/* ================================================================================================
 *  DSparseCSRMatrix (CSR SPARSE MATRIX)
 * ================================================================================================ */
TEMPLATE_WITH_TYPE_T
class DSparseCSRMatrix {
public:
    /* Metadata */
    size_t m_numRows = 0;  ///< Number of rows
    size_t m_numCols = 0;  ///< Number of columns
    size_t m_numNonZeros = 0;  ///< Number of nonzero elements

    /* Data */
    T *m_d_data = nullptr;  ///< Pointer to device data
    int *m_d_csrOffsets = nullptr;
    int *m_d_csrColumns = nullptr;
    hipsparseSpMatDescr_t m_csrMat;

    /* Buffer */
    size_t m_bufferSize = 0;
    void *m_buffer = nullptr;

public:
    DSparseCSRMatrix(const std::vector<T> &data,
                     const std::vector<int> &csrOffsets,
                     const std::vector<int> &csrColumns,
                     size_t nRows,
                     size_t nCols,
                     size_t nNonzero) :
            m_numCols(nCols), m_numRows(nRows), m_numNonZeros(nNonzero) {
        /* allocate memory */
        hipMalloc((void **) &m_d_data, m_numNonZeros * sizeof(T));
        hipMalloc((void **) &m_d_csrOffsets, (m_numRows + 1) * sizeof(int));
        hipMalloc((void **) &m_d_csrColumns, m_numNonZeros * sizeof(int));
        /* copy data to device */
        gpuErrChk(hipMemcpy(m_d_data, data.data(), m_numNonZeros * sizeof(T), hipMemcpyHostToDevice));
        gpuErrChk(hipMemcpy(m_d_csrOffsets, csrOffsets.data(), (m_numRows + 1) * sizeof(int), hipMemcpyHostToDevice));
        gpuErrChk(hipMemcpy(m_d_csrColumns, csrColumns.data(), m_numNonZeros * sizeof(int), hipMemcpyHostToDevice));
        /* create CSR */
        gpuErrChk(hipsparseCreateCsr(&m_csrMat, m_numRows, m_numCols, m_numNonZeros,
                                    m_d_csrOffsets, m_d_csrColumns, m_d_data,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    }

    ~DSparseCSRMatrix() {
        if (m_d_csrOffsets) {
            hipFree(m_d_csrOffsets);
            m_d_csrOffsets = nullptr;
        }
        if (m_d_csrColumns) {
            hipFree(m_d_csrColumns);
            m_d_csrColumns = nullptr;
        }
        if (m_d_data) {
            hipFree(m_d_data);
            m_d_data = nullptr;
        }
        if (m_buffer) {
            hipFree(m_buffer);
            m_buffer = nullptr;
        }
    }

    /**
     * Performs y = alpha * A * x + beta * y
     * @param y vector
     * @param x vector
     * @param alpha scalar
     * @param beta scalar
     */
    void axpby(hipsparseDnVecDescr_t &y,
               hipsparseDnVecDescr_t &x,
               T alpha = 1.,
               T beta = 0) {
        gpuErrChk(hipsparseSpMV_bufferSize(
                Session::getInstance().cuSpraseHandle(),
                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                &alpha, m_csrMat, x, &beta, y, HIP_R_32F,
                HIPSPARSE_SPMV_ALG_DEFAULT, &m_bufferSize));
        if (!m_buffer) {
            gpuErrChk(hipMalloc((void **) &m_buffer, m_bufferSize));
            std::cout << "m_bufferSize = " << m_bufferSize << std::endl;
        }
        float a = 1, b = 0;
        gpuErrChk(hipsparseSpMV(Session::getInstance().cuSpraseHandle(),
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &a, m_csrMat, x, &b, y, HIP_R_32F,
                               HIPSPARSE_SPMV_ALG_DEFAULT, m_buffer));
    }

};


//* ================================================================================================
// *  MAIN function (for testing only)
// * ================================================================================================ */

int main(void) {
    // MATRIX A DATA (CSR)
    const int A_num_rows = 4;
    const int A_num_cols = 4;
    const int A_nnz = 9;
    std::vector<int> hA_csrOffsets{0, 3, 4, 7, 9};
    std::vector<int> hA_columns{0, 2, 3, 1, 0, 2, 3, 1, 3};
    std::vector<float>  hA_values{1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f};

    // VECTORS
    float hX[] = {1.0f, 2.0f, 3.0f, 4.0f};
    float hY[] = {0.0f, 0.0f, 0.0f, 0.0f};
    float hY_result[] = {19.0f, 8.0f, 51.0f, 52.0f};

    // CSR object
    DSparseCSRMatrix<float> aCSR(hA_values,
                                 std::vector<int>{0, 3, 4, 7, 9},
                                 std::vector<int>{0, 2, 3, 1, 0, 2, 3, 1, 3},
                                 A_num_rows, A_num_cols, A_nnz);

    // VECTORS X and Y
    float  *dX, *dY;
    gpuErrChk(hipMalloc((void **) &dX, A_num_cols * sizeof(float)));
    gpuErrChk(hipMalloc((void **) &dY, A_num_rows * sizeof(float)));
    gpuErrChk(hipMemcpy(dX, hX, A_num_cols * sizeof(float), hipMemcpyHostToDevice));
    gpuErrChk(hipMemcpy(dY, hY, A_num_rows * sizeof(float), hipMemcpyHostToDevice));
    hipsparseDnVecDescr_t vecX, vecY;

    // Create dense vectors X and Y
    gpuErrChk(hipsparseCreateDnVec(&vecX, A_num_cols, dX, HIP_R_32F));
    gpuErrChk(hipsparseCreateDnVec(&vecY, A_num_rows, dY, HIP_R_32F));

    // execute SpMV
    aCSR.axpby(vecY, vecX);

    // destroy matrix/vector descriptors
    gpuErrChk(hipsparseDestroyDnVec(vecX));
    gpuErrChk(hipsparseDestroyDnVec(vecY));


    //--------------------------------------------------------------------------
    // device result check
    gpuErrChk(hipMemcpy(hY, dY, A_num_rows * sizeof(float), hipMemcpyDeviceToHost));
    int correct = 1;
    for (int i = 0; i < A_num_rows; i++) {
        std::cout << hY[i] << std::endl;
        if (hY[i] != hY_result[i]) { // direct floating point comparison is not
            correct = 0;             // reliable
            break;
        }
    }
    if (correct)
        printf("spmv_csr_example test PASSED\n");
    else
        printf("spmv_csr_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    gpuErrChk(hipFree(dX));
    gpuErrChk(hipFree(dY));

    return EXIT_SUCCESS;
}