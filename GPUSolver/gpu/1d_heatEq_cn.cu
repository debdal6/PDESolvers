#include <iostream>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

// Constants
#define LENGTH_OF_ROD 10.0f
#define MAX_TIME 1.0f
#define DIFFUSIVITY 1.0f
#define NUM_POINTS_SPACE 1000
#define NUM_POINTS_TIME 2000

int main() {
    // Initialize cuSPARSE
    hipsparseHandle_t cusparseHandle;
    hipsparseCreate(&cusparseHandle);

    // Calculate step sizes
    float dx = LENGTH_OF_ROD / (NUM_POINTS_SPACE - 1);
    float dt = MAX_TIME / (NUM_POINTS_TIME - 1);
    float lambda = (DIFFUSIVITY * dt) / (2.0f * dx * dx);

    // Matrix dimensions (for interior points)
    int n = NUM_POINTS_SPACE - 2;

    // Allocate host arrays
    float *h_diag = new float[n];
    float *h_upper = new float[n-1];
    float *h_lower = new float[n-1];
    float *h_temp = new float[NUM_POINTS_SPACE];
    float *h_rhs = new float[n];

    // Initialize tridiagonal matrix elements
    for(int i = 0; i < n; i++) {
        h_diag[i] = 1.0f + 2.0f * lambda;
        if(i < n-1) {
            h_upper[i] = -lambda;
            h_lower[i] = -lambda;
        }
    }

    // Initialize initial condition: u0(x) = sin(x)
    for (int i = 0; i < NUM_POINTS_SPACE; i++) {
        float x = i * dx;
        h_temp[i] = sin(x);
    }

    // Allocate device memory
    float *d_diag, *d_upper, *d_lower, *d_temp, *d_rhs;
    hipMalloc(&d_diag, n * sizeof(float));
    hipMalloc(&d_upper, (n-1) * sizeof(float));
    hipMalloc(&d_lower, (n-1) * sizeof(float));
    hipMalloc(&d_temp, NUM_POINTS_SPACE * sizeof(float));
    hipMalloc(&d_rhs, n * sizeof(float));

    // Copy data to device
    hipMemcpy(d_diag, h_diag, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_upper, h_upper, (n-1) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_lower, h_lower, (n-1) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_temp, h_temp, NUM_POINTS_SPACE * sizeof(float), hipMemcpyHostToDevice);

    // Create description for tridiagonal matrix
    hipsparseMatDescr_t descr;
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    // Create sparse matrix in CSR format
    int nnz = 3 * n - 2; // Number of non-zero elements
    int *d_csrRowPtr, *d_csrColInd;
    float *d_csrVal;
    hipMalloc(&d_csrRowPtr, (n + 1) * sizeof(int));
    hipMalloc(&d_csrColInd, nnz * sizeof(int));
    hipMalloc(&d_csrVal, nnz * sizeof(float));

    // Initialize CSR format on host
    int *h_csrRowPtr = new int[n + 1];
    int *h_csrColInd = new int[nnz];
    float *h_csrVal = new float[nnz];

    int idx = 0;
    h_csrRowPtr[0] = 0;
    for(int i = 0; i < n; i++) {
        if(i > 0) {
            h_csrColInd[idx] = i-1;
            h_csrVal[idx++] = -lambda;
        }
        h_csrColInd[idx] = i;
        h_csrVal[idx++] = 1.0f + 2.0f * lambda;
        if(i < n-1) {
            h_csrColInd[idx] = i+1;
            h_csrVal[idx++] = -lambda;
        }
        h_csrRowPtr[i+1] = idx;
    }

    // Copy CSR data to device
    hipMemcpy(d_csrRowPtr, h_csrRowPtr, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrColInd, h_csrColInd, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrVal, h_csrVal, nnz * sizeof(float), hipMemcpyHostToDevice);

    // Create sparse matrix descriptor
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void* d_buffer = nullptr;
    size_t bufferSize = 0;
    hipsparseSpSVDescr_t spsvDescr;

    // Create sparse matrix descriptor
    hipsparseCreateCsr(&matA, n, n, nnz,
                      d_csrRowPtr, d_csrColInd, d_csrVal,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // Time stepping loop
    for(int t = 1; t < NUM_POINTS_TIME; t++) {
        float t_val = t * dt;
        float alpha = 5.0f * t_val;  // This is the boundary condition alpha
        float beta = sin(LENGTH_OF_ROD) + 2.0f * t_val;

        hipMemcpy(&d_temp[0], &alpha, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&d_temp[NUM_POINTS_SPACE-1], &beta, sizeof(float), hipMemcpyHostToDevice);

        // Create vector descriptors
        hipsparseCreateDnVec(&vecX, n, d_rhs, HIP_R_32F);
        hipsparseCreateDnVec(&vecY, n, d_temp + 1, HIP_R_32F);

        // Create SpSV descriptor
        hipsparseSpSV_createDescr(&spsvDescr);

        // Prepare RHS
        float *d_temp_prev = d_temp + 1;
        hipMemcpy(d_rhs, d_temp_prev, n * sizeof(float), hipMemcpyDeviceToDevice);

        // Get buffer size and allocate
        float solve_alpha = 1.0f;  // Renamed from alpha to solve_alpha
        hipsparseSpSV_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &solve_alpha, matA, vecX, vecY, HIP_R_32F,
                               HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr, &bufferSize);
        if (d_buffer == nullptr) {
            hipMalloc(&d_buffer, bufferSize);
        }

        // Analysis phase
        hipsparseSpSV_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             &solve_alpha, matA, vecX, vecY, HIP_R_32F,
                             HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr, d_buffer);

        // Solve phase
        hipsparseSpSV_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                          &solve_alpha, matA, vecX, vecY, HIP_R_32F,
                          HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescr);

        // Cleanup vector descriptors
        hipsparseDestroyDnVec(vecX);
        hipsparseDestroyDnVec(vecY);
        hipsparseSpSV_destroyDescr(spsvDescr);

        if(t % 100 == 0) {
            hipMemcpy(h_temp, d_temp, NUM_POINTS_SPACE * sizeof(float), hipMemcpyDeviceToHost);
            printf("Time step %d completed\n", t);
        }
    }

    // Open output file
    FILE* outputFile = fopen("temperature_data.csv", "w");
    fprintf(outputFile, "x,t,temperature\n");

    // Write final state to file
    hipMemcpy(h_temp, d_temp, NUM_POINTS_SPACE * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < NUM_POINTS_SPACE; i++) {
        float x = i * dx;
        float t = MAX_TIME;  // Final time
        fprintf(outputFile, "%f,%f,%f\n", x, t, h_temp[i]);
    }

    fclose(outputFile);
    printf("Results written to temperature_data.csv\n");

    // Clean up
    hipsparseDestroy(cusparseHandle);
    hipsparseDestroyMatDescr(descr);

    hipFree(d_diag);
    hipFree(d_upper);
    hipFree(d_lower);
    hipFree(d_temp);
    hipFree(d_rhs);

    delete[] h_diag;
    delete[] h_upper;
    delete[] h_lower;
    delete[] h_temp;
    delete[] h_rhs;

    hipsparseDestroySpMat(matA);
    hipFree(d_buffer);
    hipFree(d_csrRowPtr);
    hipFree(d_csrColInd);
    hipFree(d_csrVal);
    delete[] h_csrRowPtr;
    delete[] h_csrColInd;
    delete[] h_csrVal;

    return 0;
}
